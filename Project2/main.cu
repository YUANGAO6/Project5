#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include "mnist.h"
#include "layer.h"
#include "util.h"

#include <iostream>
#include <fstream>

using namespace std;

int main() {
	ifstream train_images_is("E:/study/ECE277/proj4/ppNew Compressed (zipped) Folder/Project2/Data/train-images.idx3-ubyte", std::ios::in | std::ios::binary);
	ifstream train_labels_is("E:/study/ECE277/proj4/ppNew Compressed (zipped) Folder/Project2/Data/train-labels.idx1-ubyte", std::ios::in | std::ios::binary);
	MNIST train_dataset(train_images_is, train_labels_is);
	train_dataset.test(200, 28, 28, false);

	Layer full_connected(28, 28);
	full_connected.optimize(train_dataset, 1000, 1e-3f);

	ifstream test_images_is("E:/study/ECE277/proj4/ppNew Compressed (zipped) Folder/Project2/Data/t10k-images.idx3-ubyte", std::ios::in | std::ios::binary);
	ifstream test_labels_is("E:/study/ECE277/proj4/ppNew Compressed (zipped) Folder/Project2/Data/t10k-labels.idx1-ubyte", std::ios::in | std::ios::binary);
	MNIST test_dataset(test_images_is, test_labels_is);
	test_dataset.test(200, 28, 28, false);

	auto d_y_pred = full_connected.predict(test_dataset);
	auto h_y_pred = allocateOnHost<float>(test_dataset.m);
	hipMemcpy(h_y_pred.get(), d_y_pred.get(), test_dataset.m * sizeof(float), hipMemcpyDeviceToHost);

	int correct = 0;
	float threshold = 0.2f;
	for (size_t k = 0; k < test_dataset.m; k++) {
		bool flg = false;
		if ((test_dataset.h_y.get()[k] == 1 && h_y_pred.get()[k] >= 0.5f) || (test_dataset.h_y.get()[k] == 0 && h_y_pred.get()[k] < 0.5f)) {
			++correct;
			flg = true;
		}
		// cout << test_dataset.h_y.get()[k] << " " << h_y_pred.get()[k] << " " << flg << endl;
	}
	float error = 1.0 - ((float)correct / (float)test_dataset.m);
	cout << correct<< " to " <<  test_dataset.m << endl;
	cout << error << endl;
	getchar();

	return 0;
}